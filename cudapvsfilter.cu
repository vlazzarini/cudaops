#include "hip/hip_runtime.h"
// -*- c++ -*-
/*
  experimental cuda opcodes

  (c) Andrea Crespi, Victor Lazzarini, 2016

  This file is part of Csound.

  The Csound Library is free software; you can redistribute it
  and/or modify it under the terms of the GNU Lesser General Public
  License as published by the Free Software Foundation; either
  version 2.1 of the License, or (at your option) any later version.

  Csound is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU Lesser General Public License for more details.

  You should have received a copy of the GNU Lesser General Public
  License along with Csound; if not, write to the Free Software
  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA
  02111-1307 USA
*/

#include <csdl.h>
#include <pstream.h>

typedef struct _cudapvsfilter {
  OPDS    h;
  PVSDAT  *fout;
  PVSDAT  *fin;
  PVSDAT  *fmask;
  MYFLT   *kdepth;
  MYFLT   *igain; 
  float*  deviceInput;   // pointer to device memory (input frame)
  float*  deviceOutput;   // pointer to device memory (output frame)
  float*  deviceMask;   // pointer to device memory (mask frame)
  int  gridSize;   // number of blocks in the grid (1D)
  int blockSize;   // number of threads in one block (1D)
  uint32  lastframe;
} CUDAPVSFILTER;

// kernel for filtering in the frequency domain, given a spectral mask
__global__ void filter(float* input, float* output, float* mask, MYFLT wet, MYFLT dry, float g, int length) {
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = i<<1;
  if (j < length) {
    output[j] = (float) (input[j]*(dry+mask[j]*wet))*g;
    output[j+1] = input[j+1];
  }
} 

static int fsigs_equal(const PVSDAT *f1, const PVSDAT *f2)
{
    if (
        (f1->sliding == f2->sliding) &&
        (f1->overlap == f2->overlap) &&
        (f1->winsize == f2->winsize) &&
        (f1->wintype == f2->wintype) &&     /* harsh, maybe... */
        (f1->N == f2->N) &&
        (f1->format == f2->format))

      return 1;
    return 0;
}

static int free_device(CSOUND* csound, void* pp){
  CUDAPVSFILTER* p = (CUDAPVSFILTER*) pp;
  hipFree(p->deviceInput);
  hipFree(p->deviceOutput);
  hipFree(p->deviceMask);
  return OK;
} 

static void handleCudaError (CSOUND *csound, hipError_t error) {
  if (error!= hipSuccess) {
    csound->Message(csound, "%s in %s at line %d\n", hipGetErrorString(error),__FILE__,
       __LINE__);
    exit(EXIT_FAILURE);
  }
} 

static int cudapvsfilterset(CSOUND *csound, CUDAPVSFILTER *p)
{
  int N = p->fin->N;
  int size = (N+2) * sizeof(float);
  int maxBlockDim;
  int SMcount;
  int totNumThreads = (N+2)>>1;   // TO BE MODIFIED, MAYBE NOT
  hipError_t error;

  // get info about device
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp,0);
  maxBlockDim = deviceProp.maxThreadsPerBlock;
  SMcount = deviceProp.multiProcessorCount;
  csound->Message(csound, "cudapvsfilter running on device %s (capability %d.%d)\n", deviceProp.name,
     deviceProp.major, deviceProp.minor);
  
  // device memory allocation (input data) 
  error = hipMalloc(&p->deviceInput, size);
  handleCudaError(csound, error);
  
  // device memory allocation (input mask) 
  error = hipMalloc(&p->deviceMask, size);
  handleCudaError(csound, error);

  // device memory allocation and set to zero (output data to be computed)
  error = hipMalloc(&p->deviceOutput, size);
  handleCudaError(csound, error);
  hipMemset(p->deviceOutput,0,size);

  if (UNLIKELY(p->fin == p->fout || p->fmask == p->fout))
    csound->Warning(csound, Str("Unsafe to have same fsig as in (or filter) and out"));
  if (UNLIKELY(!(p->fout->format == PVS_AMP_FREQ) || (p->fout->format == PVS_AMP_PHASE)))
    return csound->InitError(csound, Str("cudapvsfilter: signal format "
                                           "must be amp-phase or amp-freq."));
  p->fout->sliding = 0;
  
  if (p->fout->frame.auxp == NULL || p->fout->frame.size < sizeof(float) * (N + 2))
    csound->AuxAlloc(csound, sizeof(float) * (N + 2), &p->fout->frame);
  
  p->blockSize = (((totNumThreads/SMcount)/32)+1)*32;
  if (p->blockSize > maxBlockDim) p->blockSize = maxBlockDim;
  p->gridSize = totNumThreads / p->blockSize + 1;  
  p->fout->N = N;
  p->fout->overlap = p->fin->overlap;
  p->fout->winsize = p->fin->winsize;
  p->fout->wintype = p->fin->wintype;
  p->fout->format = p->fin->format;
  p->fout->framecount = 1;
  *p->igain = 1.0;
  p->lastframe = 0;
  
  csound->RegisterDeinitCallback(csound, p, free_device);

  return OK;
}

static int cudapvsfilter(CSOUND *csound, CUDAPVSFILTER *p) {
  int N = p->fout->N;
  int framelength = N+2;
  int size = framelength * sizeof(float);
  float* fout = (float*) p->fout->frame.auxp; 
  float* fin = (float*) p->fin->frame.auxp;
  float* fmask = (float*) p->fmask->frame.auxp;
  MYFLT depth = *p->kdepth;
  float gain = (float) *p->igain;
  MYFLT dirgain;

  if (UNLIKELY(fout == NULL)) goto err1;
  if (UNLIKELY(!fsigs_equal(p->fin,p->fmask))) goto err2;
  
  if (p->lastframe < p->fin->framecount) {
    
    hipMemcpy(p->deviceInput,fin,size,hipMemcpyHostToDevice);
    hipMemcpy(p->deviceMask,fmask,size,hipMemcpyHostToDevice);
    
    depth = depth >= 0 ? (depth <= 1 ? depth : 1) : FL(0.0);   // clip depth between zero and one
    dirgain = (1 - depth);
    
    // filtering on the GPU:
    filter<<<p->gridSize,p->blockSize>>>(p->deviceInput, p->deviceOutput, p->deviceMask, depth, dirgain, gain, framelength);
    
    hipMemcpy(fout,p->deviceOutput,size,hipMemcpyDeviceToHost);

    p->fout->framecount = p->lastframe = p->fin->framecount; 
  } 
 
  return OK;

  err1: return csound->PerfError(csound, p->h.insdshead, Str("cudapvsfilter: not initialised"));
  err2: return csound->PerfError(csound, p->h.insdshead, Str("cudapvsfilter: formats are different."));
}

static OENTRY localops[] = {
  {"cudapvsfilter", sizeof(CUDAPVSFILTER),0, 3, "f", "ffxp", (SUBR) cudapvsfilterset,
   (SUBR) cudapvsfilter},
};

extern "C" {
  LINKAGE
}

