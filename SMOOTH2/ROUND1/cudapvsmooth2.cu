#include "hip/hip_runtime.h"
/* cudapvsmooth2 */

#include <csdl.h>
#include <pstream.h>

static void AuxCudaAlloc(int size, AUXCH *p){
  float *mem;
  hipMalloc(&mem, size);
  hipMemset(mem, 0, size);  
  p->auxp = mem;
  p->size = size;
}

typedef struct _cudapvsmooth2 {
  OPDS    h;
  PVSDAT  *fout;
  PVSDAT  *fin;
  MYFLT   *kfra;
  MYFLT   *kfrf; 
  AUXCH   del;
  int  gridSize;   // number of blocks in the grid (1D)
  int  blockSize;   // number of threads in one block (1D)
  uint32  lastframe;
} CUDAPVSMOOTH2;

// kernel for smoothing the time evolution functions of each channel (both amp and freq)
__global__ void smoothing(float* input, float* output, double alpha, double beta, int length) {
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = i<<1;
  if (j < length) {
    output[j] = (float) (input[j] * (1.0 + alpha) - output[j] * alpha);
    output[j+1] = (float) (input[j+1] * (1.0 + beta) - output[j+1] * beta);
  }
} 

static int free_device(CSOUND* csound, void* pp){
  CUDAPVSMOOTH2* p = (CUDAPVSMOOTH2*) pp;
  hipFree(p->fout->frame.auxp);
  return OK;
} 

static int cudapvsmooth2set(CSOUND *csound, CUDAPVSMOOTH2 *p)
{
  int N = p->fin->N;
  int size = (N+2) * sizeof(float);
  int maxBlockDim;
  int SMcount;
  int totNumThreads = (N+2)>>1;   // TO BE MODIFIED, MAYBE NOT

  // get info about device
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp,0);
  maxBlockDim = deviceProp.maxThreadsPerBlock;
  SMcount = deviceProp.multiProcessorCount;
  csound->Message(csound, "cudapvsmooth2 running on device %s (capability %d.%d)\n", deviceProp.name,
     deviceProp.major, deviceProp.minor);

  if (UNLIKELY(p->fin == p->fout))
      csound->Warning(csound, Str("Unsafe to have same fsig as in and out"));
  p->fout->NB = (N/2)+1;
  p->fout->sliding = 0;
  if (p->fout->frame.auxp == NULL || p->fout->frame.size < sizeof(float) * (N + 2))
    AuxCudaAlloc(size, &p->fout->frame);
  
  p->blockSize = (((totNumThreads/SMcount)/32)+1)*32;
  if (p->blockSize > maxBlockDim) p->blockSize = maxBlockDim;
  p->gridSize = totNumThreads / p->blockSize + 1;  
  p->fout->N = N;
  p->fout->overlap = p->fin->overlap;
  p->fout->winsize = p->fin->winsize;
  p->fout->wintype = p->fin->wintype;
  p->fout->format = p->fin->format;
  p->fout->framecount = 1;
  p->lastframe = 0;

  csound->RegisterDeinitCallback(csound, p, free_device);

  return OK;
}

static int cudapvsmooth2(CSOUND *csound, CUDAPVSMOOTH2 *p) {
  int N = p->fout->N;
  int framelength = N+2;
  double ffa = (double) *p->kfra;  // cutoff frequency in fractions of PI (for the amplitude stream)
  double ffr = (double) *p->kfrf;  // cutoff frequency in fractions of PI (for the frequency stream)

  if (p->lastframe < p->fin->framecount) {
    float   *fout, *fin;
    double  costh1, costh2, coef1, coef2;
    fout = (float *) p->fout->frame.auxp;
    fin = (float *) p->fin->frame.auxp;

    ffa = ffa < FL(0.0) ? FL(0.0) : (ffa > FL(1.0) ? FL(1.0) : ffa);
    ffr = ffr < FL(0.0) ? FL(0.0) : (ffr > FL(1.0) ? FL(1.0) : ffr);
    costh1 = 2.0 - cos(PI * ffa);
    costh2 = 2.0 - cos(PI * ffr);
    coef1 = sqrt(costh1 * costh1 - 1.0) - costh1;
    coef2 = sqrt(costh2 * costh2 - 1.0) - costh2;
    
    // channel by channel parallel filtering on the GPU (both amp and freq):
    smoothing<<<p->gridSize,p->blockSize>>>(fin, fout, coef1, coef2, framelength);

    p->fout->framecount = p->lastframe = p->fin->framecount; 
  } 
 
  return OK;
 }


static OENTRY localops[] = {
  {"cudapvsmooth2", sizeof(CUDAPVSMOOTH2),0, 3, "f", "fxx", (SUBR) cudapvsmooth2set,
   (SUBR) cudapvsmooth2, NULL}
};

extern "C" {
  LINKAGE
}

